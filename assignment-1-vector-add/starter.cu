#include "hip/hip_runtime.h"
#include <wb.h>

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)

void printme(float *in1, float* in2, float *in3, int len)
{
	for (int i = 0; i < len; i++)
	{
		wbLog(TRACE, in1[i], " ",in2[i] ," " ,in3[i]);
	}
}



__global__ void vecAdd(float *in1, float *in2, float *out, int len) {
  //@@ Insert code to implement vector addition here
	//return; 
        for (int i = (blockIdx.x * blockDim.x) + threadIdx.x; i < len; i+= blockDim.x)
	{
		out[i] = in1[i] + in2[i];
	}
}

int main(int argc, char **argv) {
  wbArg_t args;
  int inputLength;
  float *hostInput1;
  float *hostInput2;
  float *hostOutput;
  float *deviceInput1;
  float *deviceInput2;
  float *deviceOutput;

  args = wbArg_read(argc, argv);

  wbTime_start(Generic, "Importing data and creating memory on host");
  hostInput1 =
      (float *)wbImport(wbArg_getInputFile(args, 0), &inputLength);
  hostInput2 =
      (float *)wbImport(wbArg_getInputFile(args, 1), &inputLength);
  hostOutput = (float *)malloc(inputLength * sizeof(float));
  wbTime_stop(Generic, "Importing data and creating memory on host");

  wbLog(TRACE, "The input length is ", inputLength);

  wbTime_start(GPU, "Allocating GPU memory.");
  //@@ Allocate GPU memory here
	hipMalloc(&deviceInput1,(int)(inputLength * sizeof(float)));
	hipMalloc(&deviceInput2,(int) (sizeof(float) *  inputLength));
	hipMalloc(&deviceOutput,(int)(sizeof(float) *  inputLength));	

 wbLog(TRACE, "DONE GPU MEM ALLOC");
  wbTime_stop(GPU, "Allocating GPU memory.");

  wbTime_start(GPU, "Copying input memory to the GPU.");
  //@@ Copy memory to the GPU here
	hipMemcpy(deviceInput1, hostInput1, sizeof(float) * inputLength, hipMemcpyHostToDevice);
	hipMemcpy(deviceInput2, hostInput2, sizeof(float) * inputLength, hipMemcpyHostToDevice);

  wbTime_stop(GPU, "Copying input memory to the GPU.");

  //@@ Initialize the grid and block dimensions here
	int num_threads = 32;
	int num_blocks = 2;	

  wbTime_start(Compute, "Performing CUDA computation");
  //@@ Launch the GPU Kernel here
	vecAdd<<<num_threads, num_blocks>>>(deviceInput1,deviceInput2, deviceOutput, inputLength);

  hipDeviceSynchronize();
  wbTime_stop(Compute, "Performing CUDA computation");

	wbLog(TRACE,"Done running kernel atleast");
  wbTime_start(Copy, "Copying output memory to the CPU");
  //@@ Copy the GPU memory back to the CPU here
	hipMemcpy(hostOutput, deviceOutput, sizeof(float) * inputLength, hipMemcpyDeviceToHost);
  wbTime_stop(Copy, "Copying output memory to the CPU");

  wbTime_start(GPU, "Freeing GPU Memory");
  //@@ Free the GPU memory here
	hipFree(deviceInput1);
	hipFree(deviceInput2);
	hipFree(deviceOutput);
  wbTime_stop(GPU, "Freeing GPU Memory");


	printme(hostInput1, hostInput2, hostOutput, inputLength);
  wbSolution(args, hostOutput, inputLength);

  free(hostInput1);
  free(hostInput2);
  free(hostOutput);

  return 0;
}
