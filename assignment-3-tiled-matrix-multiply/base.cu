#include "hip/hip_runtime.h"

#include <wb.h>

using namespace std;

#define wbCheck(stmt)                                                     \\
  do {                                                                    \\
    hipError_t err = stmt;                                               \\
    if (err != hipSuccess) {                                             \\
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \\
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \\
      return -1;                                                          \\
    }                                                                     \\
  } while (0)

// Compute C = A * B
__global__ void matrixMultiply(float *A, float *B, float *C, int numARows,
                               int numAColumns, int numBRows,
                               int numBColumns, int numCRows,
                               int numCColumns) {
  //@@ Insert code to implement matrix multiplication here

  for (int i = (blockIdx.x * blockDim.x) + threadIdx.x; i < numCRows * numCColumns; i += blockDim.x) {
    int aindex = ((int)(i / numCColumns)) * numAColumns;
    int bindex = (i % numCColumns); 
    float sum = 0;

    for (int j = 0; j < numAColumns; j++) {
      sum += A[aindex] * B[bindex];
      aindex += 1;
      bindex += numBColumns;      
    }

    C[i] = sum;

  }
}

int main(int argc, char **argv) {
  wbArg_t args;
  float *hostA; // The A matrix
  float *hostB; // The B matrix
  float *hostC; // The output C matrix
  float *deviceA;
  float *deviceB;
  float *deviceC;
  int numARows;    // number of rows in the matrix A
  int numAColumns; // number of columns in the matrix A
  int numBRows;    // number of rows in the matrix B
  int numBColumns; // number of columns in the matrix B
  int numCRows;    // number of rows in the matrix C (you have to set this)
  int numCColumns; // number of columns in the matrix C (you have to set
                   // this)

  args = wbArg_read(argc, argv);

  wbTime_start(Generic, "Importing data and creating memory on host");
  hostA = (float *)wbImport(wbArg_getInputFile(args, 0), &numARows,
                            &numAColumns);
  hostB = (float *)wbImport(wbArg_getInputFile(args, 1), &numBRows,
                            &numBColumns);

  //@@ Set numCRows and numCColumns
  numCRows = 0;
  numCColumns = 0;
  numCRows = (numAColumns == numBRows) * numARows;
  numCColumns = numBColumns;

  if (numCRows == 0 || numCColumns == 0) {
    cerr << "Not valid C matrix" << endl;
    exit(1);
  }

  //@@ Allocate the hostC matrix
  hostC = (float*)malloc(sizeof(float) * numCRows * numCColumns); 
  //hostC = new float[numCRows * numCColumns];
  //hostC.reserve(numCRows * numCColumns);

  wbTime_stop(Generic, "Importing data and creating memory on host");
  wbLog(TRACE, "The dimensions of A are ", numARows, " x ", numAColumns);
  wbLog(TRACE, "The dimensions of B are ", numBRows, " x ", numBColumns);
  wbTime_start(GPU, "Allocating GPU memory.");

  //@@ Allocate GPU memory here
  hipMalloc(&deviceA, sizeof(float) * numARows * numAColumns);
  hipMalloc(&deviceB, sizeof(float) * numBRows * numBColumns);
  hipMalloc(&deviceC, sizeof(float) * numCRows * numCColumns);  

  wbTime_stop(GPU, "Allocating GPU memory.");
  wbTime_start(GPU, "Copying input memory to the GPU.");

  //@@ Copy memory to the GPU here
  hipMemcpy(deviceA, hostA, sizeof(float) * numARows * numAColumns, hipMemcpyHostToDevice);
  hipMemcpy(deviceB, hostB, sizeof(float) * numBRows * numBColumns, hipMemcpyHostToDevice);

  wbTime_stop(GPU, "Copying input memory to the GPU.");

  //@@ Initialize the grid and block dimensions here
  
  wbTime_start(Compute, "Performing CUDA computation");

  //@@ Launch the GPU Kernel here
  matrixMultiply<<<256,1>>>(deviceA, deviceB, deviceC,
  numARows, numAColumns, numBRows, numBColumns, numCRows, numCColumns);

  hipDeviceSynchronize();

  wbTime_stop(Compute, "Performing CUDA computation");
  wbTime_start(Copy, "Copying output memory to the CPU");

  //@@ Copy the GPU memory back to the CPU here
  hipMemcpy(hostC, deviceC, sizeof(float) * numCRows * numCColumns, hipMemcpyDeviceToHost);

  wbTime_stop(Copy, "Copying output memory to the CPU");
  wbTime_start(GPU, "Freeing GPU Memory");

  wbLog(TRACE, "C[0] IS ", hostC[0]);

  //@@ Free the GPU memory here
  hipFree(deviceA);
  hipFree(deviceB);
  hipFree(deviceC);

  wbTime_stop(GPU, "Freeing GPU Memory");
  wbSolution(args, hostC, numCRows, numCColumns);

  free(hostA);
  free(hostB);
  free(hostC);

  return 0;
}
