#include "hip/hip_runtime.h"
// Histogram Equalization

#include <wb.h>

#define HISTOGRAM_LENGTH 256
//TESTING UPDATES BITCH

//@@ insert code here

__device__ inline void atomicAdd2(float* address, float value)

{

  float old = value;  

  float new_old;

do

  {

  new_old = atomicExch(address, 0.0f);

  new_old += old;

  }

  while ((old = atomicExch(address, new_old))!=0.0f);

};

__device__
unsigned char* cast(unsigned char* outputchar, 
	float* inputfloat, 
	int imageWidth, 
	int imageHeight, 
	int imageChannels)
{
	int tidx = (blockIdx.x * blockDim.x) + threadIdx.x; 
  
	for (int i = tidx; i < imageWidth * imageHeight * imageChannels; i+= blockDim.x)
	{
		outputchar[i] = (unsigned char)(255 * inputfloat[i]);
	}
  

	return outputchar;
}

__device__
float * decast( float * outputfloat, 
	unsigned char * inputchar, 
	int imageWidth, 
	int imageHeight, 
	int imageChannels)
{
	int tidx = (blockIdx.x * blockDim.x) + threadIdx.x; 
  
	for (int i = tidx; i < imageWidth * imageHeight * imageChannels; i+= blockDim.x)
	{
		outputfloat[tidx] = (float)(inputchar[i] / 255.0);
	}
    
	return outputfloat;

}

__global__ 
void grayify(float* outputgray, 
	float* inputrgb, 
	unsigned char* inputchar,
  unsigned char* outputchar,
	int imageWidth, 
	int imageHeight, 
	int imageChannels)
{

	cast(inputchar, inputrgb, imageWidth, imageHeight, imageChannels);
	
	__syncthreads();

	int tidx = (blockIdx.x * blockDim.x) + threadIdx.x; 
  

  //ONLY 1/3 of image and 3x small images
  //Casting not working
  for (int x = tidx; x < (imageWidth * imageHeight); x += blockDim.x)
  {
    int col = (x) % imageWidth;
    int row = (x) / imageWidth;
    int ii = (row * imageWidth) + col;
    //float r = inputchar[imageChannels * ii] / 255.0 ;
    //float g = inputchar[(imageChannels * ii) + 1]/ 255.0;
    //float b = inputchar[(imageChannels * ii) + 2] / 255.0;
    unsigned char r = inputchar[imageChannels * ii];
    unsigned char g = inputchar[(imageChannels * ii) + 1];
    unsigned char b = inputchar[(imageChannels * ii) + 2] ;
    //unsigned char temp = (unsigned char)(255.0 *((unsigned char)(0.21*r) + (unsigned char)(0.71*g) + (unsigned char)(0.07*b)));
    for (int i = 0 ; i <imageChannels;i++)
    {
      outputgray[(imageChannels * ii) + i] = (float) ((0.21*r) + (0.71*g) + (0.07*b));
      outputchar[(imageChannels * ii) + i] = (unsigned char)((unsigned char)(0.21*r) + (unsigned char)(0.71*g) + (unsigned char)(0.07*b));
      /*
        output
      */
    }
    
  }
  
  /*
  for (int i = tidx; i < imageWidth * imageHeight * imageChannels; i += blockDim.x)
	{
    //TODO for (int i = 0 )
		float r = inputchar[imageChannels * i] / 255.0 ;
		float g = inputchar[(imageChannels * i) + 1]/ 255.0;
		float b = inputchar[(imageChannels * i) + 2] / 255.0;
		__syncthreads();
		unsigned char temp = (unsigned char)(255.0 *((unsigned char)(0.21*r) + (unsigned char)(0.71*g) + (unsigned char)(0.07*b)));
		outputgray[i] = (float) (temp);
		//outputchar[i] = 258;
		//outputchar[i] = temp;
		//outputchar[i] = (unsigned char)((unsigned char)(0.21*r) + (unsigned char)(0.71*g) + (unsigned char)(0.07*b));

	}
        */
        
	inputrgb = decast(inputrgb, outputchar, imageWidth, imageHeight, imageChannels);
  

}



//Use total function from list-red
__device__ 
void histify(unsigned char* inputchar, int imageWidth, int imageHeight)
{
  //unsigned char** hgram = (unsigned char**)
  //  (malloc(imageWidth * imageHeight * sizeof(unsigned char*)));

  //int idx = threadIdx.x;
  int tidx = (blockDim.x * blockIdx.x) + threadIdx.x;
  
  __shared__ float hist[256];

  for (int i = tidx; i < imageWidth * imageHeight;i += blockDim.x * gridDim.x)
  {
    //hist[inputchar[i * 3]] += 1;
    atomicAdd2(&hist[(inputchar[i * 3])], hist[(inputchar[i * 3])] += 1);
    __syncthreads();
  }

  //have mini histograms done -> test
//...

}

__device__
float p(float x, int imageWidth, int imageHeight)
{
  return x / (imageWidth * imageHeight);
}


//cdf is actually in floats but holds 256 representing characters(rgb vals)
__device__
float* calc_cdf(float* cdf, float* inputchar, int imageWidth, int imageHeight)
{
  cdf[0] = p(inputchar[0], imageWidth, imageHeight);
  for (int i = 1; i < 256; i++)
  {
    cdf[i] = cdf[i - 1] + p(inputchar[i], imageWidth, imageHeight);
  }

  return cdf;
}






int main(int argc, char **argv) 
{
  wbArg_t args;
  int imageWidth;
  int imageHeight;
  int imageChannels;
  wbImage_t inputImage;
  wbImage_t outputImage;
  float* hostInputImageData;
  float* hostOutputImageData;
  const char *inputImageFile;

  //@@ Insert more code here
  //ANY SETUP IF NEED BE??
  
  

  args = wbArg_read(argc, argv); /* parse the input arguments */

  inputImageFile = wbArg_getInputFile(args, 0);

  wbTime_start(Generic, "Importing data and creating memory on host");
  inputImage = wbImport(inputImageFile);
  imageWidth = wbImage_getWidth(inputImage);
  imageHeight = wbImage_getHeight(inputImage);
  imageChannels = wbImage_getChannels(inputImage);
  
  wbTime_stop(Generic, "Importing data and creating memory on host");

  //@@ insert code here

  //get pointers to input and output images
  hostInputImageData = (float *)malloc(imageWidth * imageHeight * imageChannels * sizeof(float));
  hostInputImageData = wbImage_getData(inputImage);
  hostOutputImageData = (float *)malloc(imageWidth * imageHeight * imageChannels * sizeof(float));

  //alloc mem and dimensions
  float* cudaInputImageData;
  float* cudaOutputImageData;
  unsigned char* cudaTempImageData;
  unsigned char* cudaTemp2ImageData;
  unsigned char* testingChar = (unsigned char*)malloc(sizeof(unsigned char) * imageHeight * imageWidth * imageChannels);
  hipMalloc(&cudaInputImageData, (int)(sizeof(float) * imageChannels * imageHeight * imageWidth));
  hipMalloc(&cudaOutputImageData, (int)(sizeof(float) * imageChannels * imageHeight * imageWidth));
  hipMalloc(&cudaTempImageData, (int)(sizeof(unsigned char) * imageChannels * imageHeight * imageWidth));
  hipMalloc(&cudaTemp2ImageData, (int)(sizeof(unsigned char) * imageChannels * imageHeight * imageWidth));
  hipMemcpy(cudaInputImageData, hostInputImageData, 
  	(int)(sizeof(float) * imageChannels * imageHeight * imageWidth), hipMemcpyHostToDevice);

  //send data to kernel
  grayify<<<256,256>>>(cudaOutputImageData, cudaInputImageData, 
  	cudaTempImageData, cudaTemp2ImageData, imageWidth, imageHeight, imageChannels);

  
  hipDeviceSynchronize();

  
  //Retrieve output image data
  hipMemcpy(hostOutputImageData, cudaOutputImageData, 
  	(sizeof(float) * imageChannels * imageHeight * imageWidth), hipMemcpyDeviceToHost);
  //for testing purps
  hipMemcpy(hostInputImageData, cudaInputImageData,
         (sizeof(float) * imageChannels * imageHeight * imageWidth), hipMemcpyDeviceToHost);
  hipMemcpy(testingChar, cudaTemp2ImageData,
         (sizeof(float) * imageChannels * imageHeight * imageWidth), hipMemcpyDeviceToHost);
  
  wbLog(TRACE, "output is ");
  for (int i = 0; i < 20; i++)
  {
	
      wbLog(TRACE,i, " ", hostInputImageData[i], " ", hostOutputImageData[i], " ", testingChar[i] );
    
  }
  
 outputImage = wbImage_new(imageWidth, imageHeight, imageChannels, hostOutputImageData);
 wbSolution(args, outputImage);

  //@@ insert code here
  hipFree(cudaInputImageData);
  hipFree(cudaOutputImageData);
  hipFree(cudaTempImageData);
  hipFree(cudaTemp2ImageData);
  free(hostInputImageData);
  free(hostOutputImageData);
  free(testingChar);  
  
  return 0;

}

