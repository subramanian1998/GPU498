#include "hip/hip_runtime.h"
// Histogram Equalization

#include <wb.h>

#define HISTOGRAM_LENGTH 256
//TESTING UPDATES BITCH

//@@ insert code here

__device__
unsigned char* cast(unsigned char* outputchar, 
	float* inputfloat, 
	int imageWidth, 
	int imageHeight, 
	int imageChannels)
{
	int tidx = (blockIdx.x * blockDim.x) + threadIdx.x; 
  
	for (int i = tidx; i < imageWidth * imageHeight * imageChannels; i+= blockDim.x)
	{
		outputchar[i] = (unsigned char)(255 * inputfloat[i]);
	}
  

	return outputchar;
}

__device__
float * decast( float * outputfloat, 
	unsigned char * inputchar, 
	int imageWidth, 
	int imageHeight, 
	int imageChannels)
{
	int tidx = (blockIdx.x * blockDim.x) + threadIdx.x; 
  
	for (int i = tidx; i < imageWidth * imageHeight * imageChannels; i+= blockDim.x)
	{
		outputfloat[tidx] = (float)(inputchar[i] / 255.0);
	}
    
	return outputfloat;

}

__global__ 
void grayify(float* outputgray, 
	float* inputrgb, 
	unsigned char* inputchar,
  unsigned char* outputchar,
	int imageWidth, 
	int imageHeight, 
	int imageChannels)
{

	cast(inputchar, inputrgb, imageWidth, imageHeight, imageChannels);
	
	__syncthreads();

	int tidx = (blockIdx.x * blockDim.x) + threadIdx.x; 
  


  for (int x = tidx; x < imageWidth * imageHeight; x += blockDim.x)
  {
    int col = x % imageWidth;
    int row = x / imageWidth;
    int ii = (row * imageWidth) + col;
    float r = inputchar[imageChannels * ii] / 255.0 ;
    float g = inputchar[(imageChannels * ii) + 1]/ 255.0;
    float b = inputchar[(imageChannels * ii) + 2] / 255.0;
    //unsigned char temp = (unsigned char)(255.0 *((unsigned char)(0.21*r) + (unsigned char)(0.71*g) + (unsigned char)(0.07*b)));
    outputgray[ii] = (float) ((0.21*r) + (0.71*g) + (0.07*b));
  }
  
  /*
  for (int i = tidx; i < imageWidth * imageHeight * imageChannels; i += blockDim.x)
	{
    //TODO for (int i = 0 )
		float r = inputchar[imageChannels * i] / 255.0 ;
		float g = inputchar[(imageChannels * i) + 1]/ 255.0;
		float b = inputchar[(imageChannels * i) + 2] / 255.0;
		__syncthreads();
		unsigned char temp = (unsigned char)(255.0 *((unsigned char)(0.21*r) + (unsigned char)(0.71*g) + (unsigned char)(0.07*b)));
		outputgray[i] = (float) (temp);
		//outputchar[i] = 258;
		//outputchar[i] = temp;
		//outputchar[i] = (unsigned char)((unsigned char)(0.21*r) + (unsigned char)(0.71*g) + (unsigned char)(0.07*b));

	}
        */
        
	//outputgray = decast(outputgray, outputchar, imageWidth, imageHeight, imageChannels);
  

}



/*
__device__ 
unsigned char** hist(unsigned char* inputchar, int imageWidth, int imageHeight)
{
  unsigned char** hgram = (unsigned char**)
    (malloc(imageWidth * imageHeight * sizeof(unsigned char*)));

  for(int i = )

}
*/




int main(int argc, char **argv) 
{
  wbArg_t args;
  int imageWidth;
  int imageHeight;
  int imageChannels;
  wbImage_t inputImage;
  wbImage_t outputImage;
  float* hostInputImageData;
  float* hostOutputImageData;
  const char *inputImageFile;

  //@@ Insert more code here
  //ANY SETUP IF NEED BE??
  
  

  args = wbArg_read(argc, argv); /* parse the input arguments */

  inputImageFile = wbArg_getInputFile(args, 0);

  wbTime_start(Generic, "Importing data and creating memory on host");
  inputImage = wbImport(inputImageFile);
  imageWidth = wbImage_getWidth(inputImage);
  imageHeight = wbImage_getHeight(inputImage);
  imageChannels = wbImage_getChannels(inputImage);
  
  wbTime_stop(Generic, "Importing data and creating memory on host");

  //@@ insert code here

  //get pointers to input and output images
  hostInputImageData = (float *)malloc(imageWidth * imageHeight * imageChannels * sizeof(float));
  hostInputImageData = wbImage_getData(inputImage);
  hostOutputImageData = (float *)malloc(imageWidth * imageHeight * imageChannels * sizeof(float));

  //alloc mem and dimensions
  float* cudaInputImageData;
  float* cudaOutputImageData;
  unsigned char* cudaTempImageData;
  unsigned char* cudaTemp2ImageData;
  hipMalloc(&cudaInputImageData, (int)(sizeof(float) * imageChannels * imageHeight * imageWidth));
  hipMalloc(&cudaOutputImageData, (int)(sizeof(float) * imageChannels * imageHeight * imageWidth));
  hipMalloc(&cudaTempImageData, (int)(sizeof(unsigned char) * imageChannels * imageHeight * imageWidth));
  hipMalloc(&cudaTemp2ImageData, (int)(sizeof(unsigned char) * imageChannels * imageHeight * imageWidth));
  hipMemcpy(cudaInputImageData, hostInputImageData, 
  	(int)(sizeof(float) * imageChannels * imageHeight * imageWidth), hipMemcpyHostToDevice);

  //send data to kernel
  grayify<<<256,256>>>(cudaOutputImageData, cudaInputImageData, 
  	cudaTempImageData, cudaTemp2ImageData, imageWidth, imageHeight, imageChannels);

  
  hipDeviceSynchronize();

  
  //Retrieve output image data
  hipMemcpy(hostOutputImageData, cudaOutputImageData, 
  	(sizeof(float) * imageChannels * imageHeight * imageWidth), hipMemcpyDeviceToHost);

  
  wbLog(TRACE, "output is ");
  for (int i = 0; i < 10; i++)
  {

      wbLog(TRACE,i, " ", hostInputImageData[i], " ", hostOutputImageData[i] );
    
  }
  
 outputImage = wbImage_new(imageWidth, imageHeight, imageChannels, hostOutputImageData);
 wbSolution(args, outputImage);

  //@@ insert code here
  hipFree(cudaInputImageData);
  hipFree(cudaOutputImageData);
  hipFree(cudaTempImageData);
  hipFree(cudaTemp2ImageData);
  free(hostInputImageData);
  free(hostOutputImageData);
  
  
  return 0;

}

