#include "hip/hip_runtime.h"
// Histogram Equalization

#include <wb.h>

#define HISTOGRAM_LENGTH 256
//TESTING UPDATES BITCH

//@@ insert code here

__device__ inline void atomicAdd2(float* address, float value)

{

  float old = value;  

  float new_old;

do

  {

  new_old = atomicExch(address, 0.0f);

  new_old += old;

  }

  while ((old = atomicExch(address, new_old))!=0.0f);

};

__device__
void cast(unsigned char* outputchar, 
	float* inputfloat, 
	int imageWidth, 
	int imageHeight, 
	int imageChannels, 
  int direction)
{
	int tidx = (blockIdx.x * blockDim.x) + threadIdx.x; 

  if (direction == 1)
  {
    for (int i = tidx; i < imageWidth * imageHeight * imageChannels; i+= blockDim.x * gridDim.x)
    {
      outputchar[i] = (unsigned char)((255 * (inputfloat[i])));
    }

    __syncthreads();
  }
  
	else {
    for (int i = tidx; i < imageWidth * imageHeight * imageChannels; i+= blockDim.x * gridDim.x)
    {
      inputfloat[i] = (float)((outputchar[i] / 255.0f));

    }
  }
}

//Use total function from list-red
__device__ 
void histify(float* globHist, unsigned char* inputchar, int imageWidth, int imageHeight)
{
  //unsigned char** hgram = (unsigned char**)
  //  (malloc(imageWidth * imageHeight * sizeof(unsigned char*)));

  //int idx = threadIdx.x;
  int tidx = (blockDim.x * blockIdx.x) + threadIdx.x;
  
  __shared__ float hist[256];


  for (int x = 0; x < gridDim.x; x++)
  {
    if (blockIdx.x == x)
    {
      for (int i = tidx; i < imageWidth * imageHeight; i += blockDim.x * gridDim.x)
      {
        //hist[inputchar[i * 3]] += 1;
        unsigned int offset = (unsigned int)((unsigned int)inputchar[i * 3]);
	      float * addr = (float *)(hist + (unsigned int)offset);
        atomicAdd((float *)(addr), (unsigned int)(1));
        //atomicAdd(&hist[(inputchar[i * 3])], hist[(inputchar[i * 3])] += 1);
        __syncthreads();
      }
      __syncthreads();
    }

   __syncthreads();
    
  }
  

  //have mini histograms done -> test -> sum upppp
  /*
  for (int i = tidx; i < 256; i+= blockDim.x * gridDim.x)
  {
    atomicAdd(&globHist[i], hist[i]);
    __syncthreads();
  }
  */

    for (int i = threadIdx.x; i < 256; i+= blockDim.x * gridDim.x)
    {
      //unsigned char offset = (unsigned char)inputchar[i * 3];
      atomicAdd((float *)(globHist + i),(unsigned int) hist[i]);
      //atomicAdd(&globHist[i], hist[i]);
      //globHist[i] = hist[i];
      __syncthreads();
    }

}

__device__
float p(unsigned char x, int imageWidth, int imageHeight)
{
  return (float)( (x * 1.0f) / (imageWidth * imageHeight));
}


//cdf is actually in floats but holds 256 representing characters(rgb vals)
__device__
float* calc_cdf(float* cdf, float* hist, int imageWidth, int imageHeight)
{
  cdf[0] = p(hist[0], imageWidth, imageHeight);
  for (int i = 1; i < 256; i++)
  {
    cdf[i] = cdf[i - 1] + p(hist[i], imageWidth, imageHeight);
  }

  return cdf;
}

__device__
unsigned char clamp(unsigned char x, unsigned char start, unsigned char end)
{
  return min(max(x, start), end);
}

__device__
unsigned char correct_val(float* cdf, unsigned char val)
{
  return clamp(255 * (cdf[val] - cdf[0]) / (1.0 - cdf[0]), 0, 255);
}

__device__
void applyhist(unsigned char * outputchar, float* cdf, int imageWidth, int imageHeight, int imageChannels)
{
  int tidx = (blockDim.x * blockIdx.x) + threadIdx.x;

  for (int i = tidx; i < imageWidth * imageHeight * imageChannels; i += blockDim.x * gridDim.x)
  {
    outputchar[i] = correct_val(cdf, outputchar[i]);
  }
}



__global__ 
void grayify(float* outputgray, 
  float* inputrgb, 
  float* hist,
  float* cdf,
  unsigned char* outputchar,
  unsigned char* inputchar,
  int imageWidth, 
  int imageHeight, 
  int imageChannels)
{

  //cast
  cast(inputchar, inputrgb, imageWidth, imageHeight, imageChannels, 1);
  
  __syncthreads();

  int tidx = (blockIdx.x * blockDim.x) + threadIdx.x; 
  
  //grayify
  for (int x = tidx; x < (imageWidth * imageHeight); x += blockDim.x)
  {
    int col = (x) % imageWidth;
    int row = (x) / imageWidth;
    int ii = (row * imageWidth) + col;

    unsigned char r = (unsigned char)(0.21 * inputchar[imageChannels * ii]);
    unsigned char g =  (unsigned char)(0.71 * inputchar[(imageChannels * ii) + 1]);
    unsigned char b = (unsigned char)(0.07 * inputchar[(imageChannels * ii) + 2]);

    __syncthreads();
    for (int i = 0 ; i <imageChannels;i++)
    {
      outputchar[(imageChannels * ii) + i] = (unsigned char)(r + g + b);
    }
  }

  //histify
  histify(hist, outputchar, imageWidth, imageHeight);

  //calc hist
  cdf = calc_cdf(cdf, hist, imageWidth, imageHeight);

  //apply hist to image
  applyhist(outputchar, cdf, imageWidth, imageHeight, imageChannels);

  //recast
  cast(outputchar, outputgray, imageWidth, imageHeight, imageChannels, 2);
  
}




int main(int argc, char **argv) 
{
  wbArg_t args;
  int imageWidth;
  int imageHeight;
  int imageChannels;
  wbImage_t inputImage;
  wbImage_t outputImage;
  float* hostInputImageData;
  float* hostOutputImageData;
  const char *inputImageFile;

  //@@ Insert more code here
  //ANY SETUP IF NEED BE??
  
  

  args = wbArg_read(argc, argv); /* parse the input arguments */

  inputImageFile = wbArg_getInputFile(args, 0);

  wbTime_start(Generic, "Importing data and creating memory on host");
  inputImage = wbImport(inputImageFile);
  imageWidth = wbImage_getWidth(inputImage);
  imageHeight = wbImage_getHeight(inputImage);
  imageChannels = wbImage_getChannels(inputImage);
  
  wbTime_stop(Generic, "Importing data and creating memory on host");

  //@@ insert code here

  //get pointers to input and output images
  hostInputImageData = (float *)malloc(imageWidth * imageHeight * imageChannels * sizeof(float));
  hostInputImageData = wbImage_getData(inputImage);
  hostOutputImageData = (float *)malloc(imageWidth * imageHeight * imageChannels * sizeof(float));
  
  //alloc mem and dimensions
  float* cudaInputImageData;
  float* cudaOutputImageData;
  unsigned char* cudaInputChar;
  unsigned char* cudaOutputChar;
  float* cudaHist;
  float* hostHist;
  hostHist = (float *)malloc(256 * sizeof(float));

  float* cudaCdf;
  hipMalloc(&cudaCdf, (sizeof(float) * 256));

  hipMalloc(&cudaInputImageData, (int)(sizeof(float) * imageChannels * imageHeight * imageWidth));
  hipMalloc(&cudaOutputImageData, (sizeof(float) * imageChannels * imageHeight * imageWidth));
  hipMalloc(&cudaHist, (sizeof(float) * 256));
  hipMalloc(&cudaInputChar, (sizeof(unsigned char) * imageChannels * imageHeight * imageWidth));
  hipMalloc(&cudaOutputChar, (sizeof(unsigned char) * imageChannels * imageHeight * imageWidth));

  hipMemcpy(cudaInputImageData, hostInputImageData, 
  	(int)(sizeof(float) * imageChannels * imageHeight * imageWidth), hipMemcpyHostToDevice);


  //send data to kernel
  grayify<<<256,256>>>(cudaOutputImageData, cudaInputImageData, cudaHist, cudaCdf, cudaOutputChar, cudaInputChar,
        imageWidth, imageHeight, imageChannels);

  
  hipDeviceSynchronize();

  
  //Retrieve output image data
  //hipMemcpy(testingChar, cudaChar,
   //      (sizeof(unsigned char) * imageChannels * imageHeight * imageWidth), hipMemcpyDeviceToHost);
  hipMemcpy(hostOutputImageData, cudaOutputImageData,
         (sizeof(float) * imageChannels * imageHeight * imageWidth), hipMemcpyDeviceToHost);
  hipMemcpy(hostHist, cudaCdf,
         (sizeof(float) * 256), hipMemcpyDeviceToHost);
  
  
  wbLog(TRACE, "output is ");
  for (int i = 0; i < 256; i++)
  {
     wbLog(TRACE, "float" , hostInputImageData[i] , " ", hostOutputImageData[i]);
    wbLog(TRACE, "hist " , hostHist[i]);
  }
  


 outputImage = wbImage_new(imageWidth, imageHeight, imageChannels, hostOutputImageData);
 wbSolution(args, outputImage);

  //@@ insert code here
  hipFree(cudaInputImageData);
  hipFree(cudaOutputChar);
  hipFree(cudaInputChar);
  hipFree(cudaHist);
  free(hostInputImageData);
  free(hostOutputImageData);
  //free(testingChar);  
  
  return 0;

}

