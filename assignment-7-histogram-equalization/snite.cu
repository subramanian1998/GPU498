#include "hip/hip_runtime.h"
// Histogram Equalization

#include <wb.h>

#define HISTOGRAM_LENGTH 256
//TESTING UPDATES BITCH

//@@ insert code here

__device__ inline void atomicAdd2(float* address, float value)

{

  float old = value;  

  float new_old;

do

  {

  new_old = atomicExch(address, 0.0f);

  new_old += old;

  }

  while ((old = atomicExch(address, new_old))!=0.0f);

};

__device__
void cast(unsigned char* outputchar, 
	float* inputfloat, 
	int imageWidth, 
	int imageHeight, 
	int imageChannels, 
  int direction)
{
	int tidx = (blockIdx.x * blockDim.x) + threadIdx.x; 

  if (direction == 1)
  {
    for (int i = tidx; i < imageWidth * imageHeight * imageChannels; i+= blockDim.x * gridDim.x)
    {
      outputchar[i] = (unsigned char)((255 * (inputfloat[i])));
    }

    __syncthreads();
  }
  
	else {
    for (int i = tidx; i < imageWidth * imageHeight * imageChannels; i+= blockDim.x * gridDim.x)
    {
      inputfloat[i] = (float)((outputchar[i] / 255.0f));

    }
  }

}

__global__ 
void grayify(float* outputgray, 
  float* inputrgb, 
  float* hist,
  unsigned char* outputchar,
  int imageWidth, 
  int imageHeight, 
  int imageChannels)
{

  cast(outputchar, inputrgb, imageWidth, imageHeight, imageChannels, 1);
  
  __syncthreads();

  int tidx = (blockIdx.x * blockDim.x) + threadIdx.x; 
  

  //ONLY 1/3 of image and 3x small images
  //Casting not working
  for (int x = tidx; x < (imageWidth * imageHeight); x += blockDim.x)
  {
    int col = (x) % imageWidth;
    int row = (x) / imageWidth;
    int ii = (row * imageWidth) + col;
    //float r = inputchar[imageChannels * ii] / 255.0 ;
    //float g = inputchar[(imageChannels * ii) + 1]/ 255.0;
    //float b = inputchar[(imageChannels * ii) + 2] / 255.0;
    unsigned char r = (unsigned char)(0.21 * inputchar[imageChannels * ii]);
    unsigned char g =  (unsigned char)(0.71 * inputchar[(imageChannels * ii) + 1]);
    unsigned char b = (unsigned char)(0.07 * inputchar[(imageChannels * ii) + 2]) ;
    //unsigned char temp = (unsigned char)(255.0 *((unsigned char)(0.21*r) + (unsigned char)(0.71*g) + (unsigned char)(0.07*b)));
    for (int i = 0 ; i <imageChannels;i++)
    {
      //outputgray[(imageChannels * ii) + i] = (float) ((0.21*r) + (0.71*g) + (0.07*b));
      outputchar[(imageChannels * ii) + i] = (unsigned char)(r + g + b);
    }
  }
        
  cast(outputchar, outputgray, imageWidth, imageHeight, imageChannels, 2);
  

}




int main(int argc, char **argv) 
{
  wbArg_t args;
  int imageWidth;
  int imageHeight;
  int imageChannels;
  wbImage_t inputImage;
  wbImage_t outputImage;
  float* hostInputImageData;
  float* hostOutputImageData;
  const char *inputImageFile;

  //@@ Insert more code here
  //ANY SETUP IF NEED BE??
  
  

  args = wbArg_read(argc, argv); /* parse the input arguments */

  inputImageFile = wbArg_getInputFile(args, 0);

  wbTime_start(Generic, "Importing data and creating memory on host");
  inputImage = wbImport(inputImageFile);
  imageWidth = wbImage_getWidth(inputImage);
  imageHeight = wbImage_getHeight(inputImage);
  imageChannels = wbImage_getChannels(inputImage);
  
  wbTime_stop(Generic, "Importing data and creating memory on host");

  //@@ insert code here

  //get pointers to input and output images
  hostInputImageData = (float *)malloc(imageWidth * imageHeight * imageChannels * sizeof(float));
  hostInputImageData = wbImage_getData(inputImage);
  hostOutputImageData = (float *)malloc(imageWidth * imageHeight * imageChannels * sizeof(float));
  
  //alloc mem and dimensions
  float* cudaInputImageData;
  float* cudaOutputImageData;
  unsigned char* cudaChar;
  float* cudaHist;
  float* hostHist;
  hostHist = (float *)malloc(256 * sizeof(float));
  //unsigned char* testingChar;
  //unsigned char* confirmedChar;
  //testingChar = (unsigned char*)malloc(sizeof(unsigned char) * imageHeight * imageWidth * imageChannels);
  //confirmedChar = (unsigned char*)malloc(sizeof(unsigned char) * imageHeight * imageWidth * imageChannels);
  hipMalloc(&cudaInputImageData, (int)(sizeof(float) * imageChannels * imageHeight * imageWidth));
  hipMalloc(&cudaOutputImageData, (sizeof(float) * imageChannels * imageHeight * imageWidth));
  hipMalloc(&cudaHist, (sizeof(float) * 256));
  hipMalloc(&cudaChar, (sizeof(unsigned char) * imageChannels * imageHeight * imageWidth));
  hipMemcpy(cudaInputImageData, hostInputImageData, 
  	(int)(sizeof(float) * imageChannels * imageHeight * imageWidth), hipMemcpyHostToDevice);

  /*
  for (int i = 0; i < imageChannels * imageHeight * imageWidth; i++ )
  {
    confirmedChar[i] = (unsigned char)(255 * (hostInputImageData[i]));
  }
  */
  //wbLog(TRACE, "output is ", testingChar[0], ' ', (unsigned char)(255 * hostInputImageData[0]) );


  //send data to kernel
  cast<<<256,256>>>(cudaOutputImageData, cudaInputImageData, cudaHist, cudaChar,
        imageWidth, imageHeight, imageChannels);

  
  hipDeviceSynchronize();

  
  //Retrieve output image data
  hipMemcpy(testingChar, cudaChar,
         (sizeof(unsigned char) * imageChannels * imageHeight * imageWidth), hipMemcpyDeviceToHost);
  hipMemcpy(hostOutputImageData, cudaOutputImageData,
         (sizeof(float) * imageChannels * imageHeight * imageWidth), hipMemcpyDeviceToHost);
  hipMemcpy(hostHist, cudaHist,
         (sizeof(float) * 256), hipMemcpyDeviceToHost);
  
  
  wbLog(TRACE, "output is ");
  for (int i = 0; i < 20; i++)
  {
	   //unsigned char temp = testingChar[i];
     //wbLog(TRACE, "char" , confirmedChar[i], " ", temp);
     wbLog(TRACE, "float" , hostInputImageData[i] , " ", hostOutputImageData[i]);
  }
  


 outputImage = wbImage_new(imageWidth, imageHeight, imageChannels, hostOutputImageData);
 wbSolution(args, outputImage);

  //@@ insert code here
  hipFree(cudaInputImageData);
  hipFree(cudaTemp2ImageData);
  free(hostInputImageData);
  free(hostOutputImageData);
  free(testingChar);  
  
  return 0;

}

