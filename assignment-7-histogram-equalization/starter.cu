#include "hip/hip_runtime.h"
// Histogram Equalization

#include <wb.h>

#define HISTOGRAM_LENGTH 256
//TESTING UPDATES BITCH

//@@ insert code here
__device__
unsigned char * cast(unsigned char * outputchar, 
	float * inputfloat, 
	int imageWidth, 
	int imageHeight, 
	int imageChannels)
{
	int tidx = (blockIdx.x * blockDim.x) + threadIdx.x; 
	for (int i = tidx; i < imageWidth * imageHeight * imageChannels; i+= blockDim.x)
	{
		outputchar[i] = (unsigned char)(255 * inputfloat[i]);
	}

	return outputchar;
}

__device__
float * decast( float * outputfloat, 
	unsigned char * inputchar, 
	int imageWidth, 
	int imageHeight, 
	int imageChannels)
{
	int tidx = (blockIdx.x * blockDim.x) + threadIdx.x; 
	for (int i = tidx; i < imageWidth * imageHeight * imageChannels; i+= blockDim.x)
	{
		outputfloat[i] = (float)(inputchar[i] / 255.0);
	}

	return outputfloat;

}

__global__ 
void grayify(float * outputgray, 
	float * inputrgb, 
	unsigned char * inputchar,
	int imageWidth, 
	int imageHeight, 
	int imageChannels)
{

	inputchar = cast(inputchar, inputrgb, imageWidth, imageHeight, imageChannels);
	
	__syncthreads();

	int tidx = (blockIdx.x * blockDim.x) + threadIdx.x; 

	for (int i = tidx; i < imageWidth * imageHeight * imageChannels; i += blockDim.x)
	{
		float r = inputchar[imageChannels * i];
		float g = inputchar[(imageChannels * i) + 1];
		float b = inputchar[(imageChannels * i) + 1];
		__syncthreads();
		inputchar[tidx] = (unsigned char) (0.21*r + 0.71*g + 0.07*b);
	}


	outputgray = decast(outputgray, inputchar, imageWidth, imageHeight, imageChannels);


}


int main(int argc, char **argv) 
{
  wbArg_t args;
  int imageWidth;
  int imageHeight;
  int imageChannels;
  wbImage_t inputImage;
  wbImage_t outputImage;
  float *hostInputImageData;
  float *hostOutputImageData;
  const char *inputImageFile;

  //@@ Insert more code here
  //ANY SETUP IF NEED BE??
  
  

  args = wbArg_read(argc, argv); /* parse the input arguments */

  inputImageFile = wbArg_getInputFile(args, 0);

  wbTime_start(Generic, "Importing data and creating memory on host");
  inputImage = wbImport(inputImageFile);
  imageWidth = wbImage_getWidth(inputImage);
  imageHeight = wbImage_getHeight(inputImage);
  imageChannels = wbImage_getChannels(inputImage);
  outputImage = wbImage_new(imageWidth, imageHeight, imageChannels);
  wbTime_stop(Generic, "Importing data and creating memory on host");

  //@@ insert code here

  //get pointers to input and output images
  hostInputImageData = wbImage_getData(inputImage);
  hostOutputImageData = wbImage_getData(outputImage);

  //alloc mem and dimensions
  float * cudaInputImageData, cudaOutputImageData;
  unsigned char * cudaTempImageData;
  hipMalloc(&cudaInputImageData, (int)(sizeof(float) * imageChannels * imageHeight * imageWidth);
  hipMalloc(&cudaOutputImageData, (int)(sizeof(float) * imageChannels * imageHeight * imageWidth));
  hipMalloc(&cudaTempImageData, (int)(sizeof(unsigned char) * imageChannels * imageHeight * imageWidth));
  hipMemcpy(cudaInputImageData, hostInputImageData, 
  	sizeof(float) * imageChannels * imageHeight * imageWidth, hipMemcpyHostToDevice);

  //send data to kernel
  grayify<<<256,256>>>(cudaOutputImageData, cudaInputImageData, 
  	cudaTempImageData, imageWidth, imageHeight, imageChannels);


  hipDeviceSynchronize();


  //Retrieve output image data
  hipMemcpy(hostOutputImageData, cudaOutputImageData, cudaTempImageData, 
  	sizeof(float) * imageChannels * imageHeight * imageWidth, hipMemcpyDeviceToHost);


  wbSolution(args, outputImage);

  //@@ insert code here

  

  return 0;

}

