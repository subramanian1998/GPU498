#include "hip/hip_runtime.h"
// Histogram Equalization

#include <wb.h>

#define HISTOGRAM_LENGTH 256
//TESTING UPDATES BITCH

//@@ insert code here
__global__
unsigned char * cast( float * outputchar, 
	float * inputfloat, 
	int imageWidth, 
	int imageHeight, 
	int imageChannels)
{
	int tidx = (blockIdx.x * blockDim.x) + threadIdx.x; 
	for (int i = tidx; i < imageWidth * imageHeight * imageChannels, i+= blockDim.x)
	{
		outputchar[i] = (unsigned char)(255 * inputfloat[i]);
	}

	return outputchar;
}

__global__ 
void grayify( float * outputgray, 
	float * inputrgb, 
	int imageWidth, 
	int imageHeight, 
	int imageChannels)
{

	outputgray = cast(outputgray, inputrgb, imageWidth, imageHeight, imageChannels);
	
	synchronize();

	int tidx = (blockIdx.x * blockDim.x) + threadIdx.x; 

	for (int i = tidx; i < imageWidth * imageHeight * imageChannels; i += blockDim.x)
	{
		r = outputgray[imageChannels * i];
		g = outputgray[(imageChannels * i) + 1];
		b = outputgray[(imageChannels * i) + 1];
		synchronize();
		outputgray[tidx] = (unsigned char) (0.21*r + 0.71*g + 0.07*b);
	}

}


int main(int argc, char **argv) {
  wbArg_t args;
  int imageWidth;
  int imageHeight;
  int imageChannels;
  wbImage_t inputImage;
  wbImage_t outputImage;
  float *hostInputImageData;
  float *hostOutputImageData;
  const char *inputImageFile;

  //@@ Insert more code here
  //ANY SETUP IF NEED BE??
  
  

  args = wbArg_read(argc, argv); /* parse the input arguments */

  inputImageFile = wbArg_getInputFile(args, 0);

  wbTime_start(Generic, "Importing data and creating memory on host");
  inputImage = wbImport(inputImageFile);
  imageWidth = wbImage_getWidth(inputImage);
  imageHeight = wbImage_getHeight(inputImage);
  imageChannels = wbImage_getChannels(inputImage);
  outputImage = wbImage_new(imageWidth, imageHeight, imageChannels);
  wbTime_stop(Generic, "Importing data and creating memory on host");

  //@@ insert code here

  //get pointers to input and output images
  hostInputImageData = wbImage_getData(inputImage);
  hostOutputImageData = wbImage_getData(outputImage);

  //alloc mem and dimensions
  float * cudaInputImageData, cudaOutputImageData;
  hipMalloc(&cudaInputImageData, sizeof(FLOAT) * imageChannels * imageHeight * imageWidth);
  hipMemcpy(cudaInputImageData, hostInputImageData, 
  	sizeof(FLOAT) * imageChannels * imageHeight * imageWidth, hipMemcpyHostToDevice);

  //send data to kernel
  grayify<<<256,256>>>(cudaOutputImageData, cudaInputImageData, 
  	imageWidth, imageHeight, imageChannels);


  hipDeviceSynchronize();


  //Retrieve output image data
  hipMemcpy(hostOutputImageData, cudaOutputImageData, 
  	sizeof(FLOAT) * imageChannels * imageHeight * imageWidth, hipMemcpyDeviceToHost);


  wbSolution(args, outputImage);

  //@@ insert code here

  


  return 0;
}
