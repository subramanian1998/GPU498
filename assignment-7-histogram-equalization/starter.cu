#include "hip/hip_runtime.h"
// Histogram Equalization

#include <wb.h>

#define HISTOGRAM_LENGTH 256
//TESTING UPDATES BITCH

//@@ insert code here

__device__
unsigned char* cast(unsigned char* outputchar, 
	float* inputfloat, 
	int imageWidth, 
	int imageHeight, 
	int imageChannels)
{
	int tidx = (blockIdx.x * blockDim.x) + threadIdx.x; 
  
	for (int i = tidx; i < imageWidth * imageHeight * imageChannels; i+= blockDim.x)
	{
		outputchar[i] = (unsigned char)(255 * inputfloat[i]);
	}
  

	return outputchar;
}

__device__
float * decast( float* outputfloat, 
	unsigned char* inputchar, 
	int imageWidth, 
	int imageHeight, 
	int imageChannels)
{
	int tidx = (blockIdx.x * blockDim.x) + threadIdx.x; 
  
	for (int i = tidx; i < imageWidth * imageHeight * imageChannels; i+= blockDim.x)
	{
		outputfloat[i] = (float)(inputchar[i] / 255.0);
	}
    
	return outputfloat;

}

__global__ 
void grayify(float* outputgray, 
	float* inputrgb, 
	unsigned char* inputchar,
  unsigned char* tempchar,
	int imageWidth, 
	int imageHeight, 
	int imageChannels)
{

	inputchar = cast(inputchar, inputrgb, imageWidth, imageHeight, imageChannels);
	
	__syncthreads();

	int tidx = (blockIdx.x * blockDim.x) + threadIdx.x; 

  //unsigned char* tempchar = (unsigned char*)malloc(sizeof(unsigned char) * imageWidth * imageHeight * imageChannels);
  //memcpy(tempchar, inputchar, sizeof(unsigned char) * imageHeight * imageChannels * imageWidth);

  for (int i = tidx; i < imageWidth * imageHeight * imageChannels; i += blockDim.x)
	{
		float r = inputchar[imageChannels * i];
		float g = inputchar[(imageChannels * i) + 1];
		float b = inputchar[(imageChannels * i) + 2];
		//tempchar[i] = (unsigned char) ((0.21*r) + (0.71*g) + (0.07*b));

	}

  __syncthreads();
  
	outputgray = decast(outputgray, inputchar, imageWidth, imageHeight, imageChannels);

}



/*
__device__ 
unsigned char** hist(unsigned char* inputchar, int imageWidth, int imageHeight)
{
  unsigned char** hgram = (unsigned char**)
    (malloc(imageWidth * imageHeight * sizeof(unsigned char*)));

  for(int i = )

}
*/




int main(int argc, char **argv) 
{
  wbArg_t args;
  int imageWidth;
  int imageHeight;
  int imageChannels;
  wbImage_t inputImage;
  wbImage_t outputImage;
  float* hostInputImageData;
  float* hostOutputImageData;
  const char *inputImageFile;

  //@@ Insert more code here
  //ANY SETUP IF NEED BE??
  
  

  args = wbArg_read(argc, argv); /* parse the input arguments */

  inputImageFile = wbArg_getInputFile(args, 0);

  wbTime_start(Generic, "Importing data and creating memory on host");
  inputImage = wbImport(inputImageFile);
  imageWidth = wbImage_getWidth(inputImage);
  imageHeight = wbImage_getHeight(inputImage);
  imageChannels = wbImage_getChannels(inputImage);
  
  wbTime_stop(Generic, "Importing data and creating memory on host");

  //@@ insert code here

  //get pointers to input and output images
  hostInputImageData = (float *)malloc(imageWidth * imageHeight * imageChannels * sizeof(float));
  hostInputImageData = wbImage_getData(inputImage);
  hostOutputImageData = (float *)malloc(imageWidth * imageHeight * imageChannels * sizeof(float));

  //alloc mem and dimensions
  float* cudaInputImageData;
  float* cudaOutputImageData;
  unsigned char* cudaTempImageData;
  unsigned char* cudaCharImageData;
  hipMalloc(&cudaInputImageData, (int)(sizeof(float) * imageChannels * imageHeight * imageWidth));
  hipMalloc(&cudaOutputImageData, (int)(sizeof(float) * imageChannels * imageHeight * imageWidth));
  hipMalloc(&cudaTempImageData, (int)(sizeof(unsigned char) * imageChannels * imageHeight * imageWidth));
  hipMalloc(&cudaCharImageData, (int)(sizeof(unsigned char) * imageChannels * imageHeight * imageWidth));
  hipMemcpy(&cudaInputImageData, hostInputImageData, 
  	(int)(sizeof(float) * imageChannels * imageHeight * imageWidth), hipMemcpyHostToDevice);

  //send data to kernel
  grayify<<<256,256>>>(cudaOutputImageData, cudaInputImageData, 
  	cudaTempImageData, cudaCharImageData, imageWidth, imageHeight, imageChannels);

  
  hipDeviceSynchronize();

  
  //Retrieve output image data
  hipMemcpy(hostOutputImageData, cudaOutputImageData, 
  	(sizeof(float) * imageChannels * imageHeight * imageWidth), hipMemcpyDeviceToHost);

  
  wbLog(TRACE, "output is ");
  /*  
  for (int i = 0; i < imageHeight * imageWidth * imageChannels; i++)
  {
    //22510
    //36010
    if (i > 36010) {
      wbLog(TRACE,i, " ", hostInputImageData[i], " ", hostOutputImageData[i] );
    }
  }
  */
 outputImage = wbImage_new(imageWidth, imageHeight, imageChannels, hostOutputImageData);
 wbSolution(args, outputImage);

  //@@ insert code here
  hipFree(cudaInputImageData);
  hipFree(cudaOutputImageData);
  hipFree(cudaTempImageData);
  hipFree(cudaCharImageData);
  free(hostInputImageData);
  free(hostOutputImageData);
  
  
  return 0;

}

