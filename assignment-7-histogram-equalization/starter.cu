#include "hip/hip_runtime.h"
// Histogram Equalization

#include <wb.h>

#define HISTOGRAM_LENGTH 256
//TESTING UPDATES BITCH

//@@ insert code here
__device__
unsigned char* cast(unsigned char* outputchar, 
	float* inputfloat, 
	int imageWidth, 
	int imageHeight, 
	int imageChannels)
{
	int tidx = (blockIdx.x * blockDim.x) + threadIdx.x; 
  /*
	for (int i = tidx; i < imageWidth * imageHeight * imageChannels; i+= blockDim.x)
	{
		outputchar[i] = (unsigned char)(255 * inputfloat[i]);
	}
  */

  outputchar[tidx] = (unsigned char)(255); //* inputfloat[tidx]);

	return outputchar;
}

__device__
float * decast( float * outputfloat, 
	unsigned char * inputchar, 
	int imageWidth, 
	int imageHeight, 
	int imageChannels)
{
	int tidx = (blockIdx.x * blockDim.x) + threadIdx.x; 
  /*
	for (int i = tidx; i < imageWidth * imageHeight * imageChannels; i+= blockDim.x)
	{
		outputfloat[tidx] = (float)(inputchar[tidx] / 255.0);
	}
    */

  outputfloat[tidx] = (float)(1.0);//inputchar[tidx] / 255.0);
	return outputfloat;

}

__global__ 
void grayify(float* outputgray, 
	float* inputrgb, 
	unsigned char* inputchar,
	int imageWidth, 
	int imageHeight, 
	int imageChannels)
{

	//cast(inputchar, inputrgb, imageWidth, imageHeight, imageChannels);
	
	__syncthreads();

	int tidx = (blockIdx.x * blockDim.x) + threadIdx.x; 
        /*  
	for (int i = tidx; i < imageWidth * imageHeight * imageChannels; i += blockDim.x)
	{
    //TODO for (int i = 0 )
		float r = inputchar[imageChannels * i];
		float g = inputchar[(imageChannels * i) + 1];
		float b = inputchar[(imageChannels * i) + 2];
		__syncthreads();
		inputchar[i] = (unsigned char) (0.21*r + 0.71*g + 0.07*b);
	}

        */
	//outputgray = decast(outputgray, inputchar, imageWidth, imageHeight, imageChannels);
        outputgray = inputrgb;

}

/*
__device__ 
unsigned char** hist(unsigned char* inputchar, int imageWidth, int imageHeight)
{
  unsigned char** hgram = (unsigned char**)
    (malloc(imageWidth * imageHeight * sizeof(unsigned char*)));

  for(int i = )

}
*/




int main(int argc, char **argv) 
{
  wbArg_t args;
  int imageWidth;
  int imageHeight;
  int imageChannels;
  wbImage_t inputImage;
  wbImage_t outputImage;
  float *hostInputImageData;
  float *hostOutputImageData;
  const char *inputImageFile;

  //@@ Insert more code here
  //ANY SETUP IF NEED BE??
  
  

  args = wbArg_read(argc, argv); /* parse the input arguments */

  inputImageFile = wbArg_getInputFile(args, 0);

  wbTime_start(Generic, "Importing data and creating memory on host");
  inputImage = wbImport(inputImageFile);
  imageWidth = wbImage_getWidth(inputImage);
  imageHeight = wbImage_getHeight(inputImage);
  imageChannels = wbImage_getChannels(inputImage);
  
  wbTime_stop(Generic, "Importing data and creating memory on host");

  //@@ insert code here

  //get pointers to input and output images
  hostInputImageData = wbImage_getData(inputImage);
  //hostOutputImageData = wbImage_getData(outputImage);

  //alloc mem and dimensions
  float* cudaInputImageData;
  float* cudaOutputImageData;
  unsigned char* cudaTempImageData;
  hipMalloc((void **)&cudaInputImageData, (int)(sizeof(float) * imageChannels * imageHeight * imageWidth));
  hipMalloc((void **)&cudaOutputImageData, (int)(sizeof(float) * imageChannels * imageHeight * imageWidth));
  hipMalloc(&cudaTempImageData, (int)(sizeof(unsigned char) * imageChannels * imageHeight * imageWidth));
  hipMemcpy(cudaInputImageData, hostInputImageData, 
  	sizeof(float) * imageChannels * imageHeight * imageWidth, hipMemcpyHostToDevice);

  //send data to kernel
  grayify<<<256,256>>>(cudaOutputImageData, cudaInputImageData, 
  	cudaTempImageData, imageWidth, imageHeight, imageChannels);


  hipDeviceSynchronize();


  //Retrieve output image data
  hipMemcpy(&hostOutputImageData, &cudaOutputImageData, 
  	(int)(sizeof(float) * imageChannels * imageHeight * imageWidth), hipMemcpyDeviceToHost);


  outputImage = wbImage_new(imageWidth, imageHeight, imageChannels, hostOutputImageData);
  wbSolution(args, outputImage);

  //@@ insert code here

  

  return 0;

}

