#include "hip/hip_runtime.h"
// Histogram Equalization

#include <wb.h>

#define HISTOGRAM_LENGTH 256
//TESTING UPDATES BITCH

//@@ insert code here
__device__
unsigned char * cast(unsigned char * outputchar, 
	float * inputfloat, 
	int imageWidth, 
	int imageHeight, 
	int imageChannels)
{
	int tidx = (blockIdx.x * blockDim.x) + threadIdx.x; 
	for (int i = tidx; i < imageWidth * imageHeight * imageChannels; i+= blockDim.x)
	{
		outputchar[i] = (unsigned char)(255 * inputfloat[i]);
	}

	return outputchar;
}

__device__
float * decast( float * outputchar, 
	unsigned char * inputfloat, 
	int imageWidth, 
	int imageHeight, 
	int imageChannels)
{
	int tidx = (blockIdx.x * blockDim.x) + threadIdx.x; 
	for (int i = tidx; i < imageWidth * imageHeight * imageChannels; i+= blockDim.x)
	{
		outputchar[i] = (float)(inputfloat[i] / 255.0);
	}

}

__global__ 
void grayify(float * outputgray, 
	float * inputrgb, 
	char * temp,
	int imageWidth, 
	int imageHeight, 
	int imageChannels)
{

	outputgray = cast(outputgray, inputrgb, imageWidth, imageHeight, imageChannels);
	
	synchronize();

	int tidx = (blockIdx.x * blockDim.x) + threadIdx.x; 

	for (int i = tidx; i < imageWidth * imageHeight * imageChannels; i += blockDim.x)
	{
		float r = outputgray[imageChannels * i];
		float g = outputgray[(imageChannels * i) + 1];
		float b = outputgray[(imageChannels * i) + 1];
		synchronize();
		outputgray[tidx] = (unsigned char) (0.21*r + 0.71*g + 0.07*b);
	}

}


int main(int argc, char **argv) {
  wbArg_t args;
  int imageWidth;
  int imageHeight;
  int imageChannels;
  wbImage_t inputImage;
  wbImage_t outputImage;
  float *hostInputImageData;
  float *hostOutputImageData;
  const char *inputImageFile;

  //@@ Insert more code here
  //ANY SETUP IF NEED BE??
  
  

  args = wbArg_read(argc, argv); /* parse the input arguments */

  inputImageFile = wbArg_getInputFile(args, 0);

  wbTime_start(Generic, "Importing data and creating memory on host");
  inputImage = wbImport(inputImageFile);
  imageWidth = wbImage_getWidth(inputImage);
  imageHeight = wbImage_getHeight(inputImage);
  imageChannels = wbImage_getChannels(inputImage);
  outputImage = wbImage_new(imageWidth, imageHeight, imageChannels);
  wbTime_stop(Generic, "Importing data and creating memory on host");

  //@@ insert code here

  //get pointers to input and output images
  hostInputImageData = wbImage_getData(inputImage);
  hostOutputImageData = wbImage_getData(outputImage);

  //alloc mem and dimensions
  float * cudaInputImageData, cudaOutputImageData;
  hipMalloc(&cudaInputImageData, sizeof(float) * imageChannels * imageHeight * imageWidth);
  hipMalloc(&cudaOutputImageData, sizeof(float) * imageChannels * imageHeight * imageWidth);
  hipMalloc(&cudaTempImageData, sizeof(unsigned char) * imageChannels * imageHeight * imageWidth);
  hipMemcpy(cudaInputImageData, hostInputImageData, 
  	sizeof(float) * imageChannels * imageHeight * imageWidth, hipMemcpyHostToDevice);

  //send data to kernel
  grayify<<<256,256>>>(cudaOutputImageData, cudaInputImageData, 
  	imageWidth, imageHeight, imageChannels);


  hipDeviceSynchronize();


  //Retrieve output image data
  hipMemcpy(hostOutputImageData, cudaOutputImageData, cudaTempImageData, 
  	sizeof(float) * imageChannels * imageHeight * imageWidth, hipMemcpyDeviceToHost);


  wbSolution(args, outputImage);

  //@@ insert code here

  


  return 0;
}
