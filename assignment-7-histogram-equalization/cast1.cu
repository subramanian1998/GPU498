#include "hip/hip_runtime.h"
// Histogram Equalization

#include <wb.h>

#define HISTOGRAM_LENGTH 256
//TESTING UPDATES BITCH

//@@ insert code here


__global__
void cast(float* outputchar, 
	float* inputfloat, 
	int imageWidth, 
	int imageHeight, 
	int imageChannels)
{
	/*	for (int i = 0; i< 1000; i++) 
{
	outputchar[i] = (unsigned char)('c');
}*/
	int tidx = (blockIdx.x * blockDim.x) + threadIdx.x; 
  
	for (int i = tidx; i < imageWidth * imageHeight * imageChannels; i+= blockDim.x * gridDim.x)
	{
	  outputchar[i] = (unsigned char)((int)(255 * (inputfloat[i])));
          //outputchar[i] = temp;
	  //outputchar[i] = 'c';
  }

  for (int i = tidx; i < imageWidth * imageHeight * imageChannels; i+= blockDim.x * gridDim.x)
  {
    inputfloat[i] = (float)((outputchar[i] / 255.0f));
          //outputchar[i] = temp;
    //outputchar[i] = 'c';
  }



}



int main(int argc, char **argv) 
{
  wbArg_t args;
  int imageWidth;
  int imageHeight;
  int imageChannels;
  wbImage_t inputImage;
  wbImage_t outputImage;
  float* hostInputImageData;
  float* hostOutputImageData;
  const char *inputImageFile;

  //@@ Insert more code here
  //ANY SETUP IF NEED BE??
  
  

  args = wbArg_read(argc, argv); /* parse the input arguments */

  inputImageFile = wbArg_getInputFile(args, 0);

  wbTime_start(Generic, "Importing data and creating memory on host");
  inputImage = wbImport(inputImageFile);
  imageWidth = wbImage_getWidth(inputImage);
  imageHeight = wbImage_getHeight(inputImage);
  imageChannels = wbImage_getChannels(inputImage);
  
  wbTime_stop(Generic, "Importing data and creating memory on host");

  //@@ insert code here

  //get pointers to input and output images
  hostInputImageData = (float *)malloc(imageWidth * imageHeight * imageChannels * sizeof(float));
  hostInputImageData = wbImage_getData(inputImage);
  //alloc mem and dimensions
  float* cudaInputImageData;
  float* cudaOutputImageData;
  float* cudaTemp2ImageData;
  unsigned char* testingChar;
  testingChar = (unsigned char*)malloc(sizeof(unsigned char) * imageHeight * imageWidth * imageChannels);
  hipMalloc(&cudaInputImageData, (int)(sizeof(float) * imageChannels * imageHeight * imageWidth));
  hipMalloc(&cudaTemp2ImageData, (sizeof(unsigned char) * imageChannels * imageHeight * imageWidth));
  hipMemcpy(cudaInputImageData, hostInputImageData, 
  	(int)(sizeof(float) * imageChannels * imageHeight * imageWidth), hipMemcpyHostToDevice);

  //send data to kernel
  imageHeight = 10;
  imageWidth = 10;
  imageChannels = 3;
  cast<<<256,256>>>(cudaTemp2ImageData, cudaInputImageData, 
        imageWidth, imageHeight, imageChannels);

  
  hipDeviceSynchronize();

  
  //Retrieve output image data
  hipMemcpy(testingChar, cudaTemp2ImageData,
         (sizeof(unsigned char) * imageChannels * imageHeight * imageWidth), hipMemcpyDeviceToHost);
  hipMemcpy(hostInputImageData, cudaInputImageData,
         (sizeof(float) * imageChannels * imageHeight * imageWidth), hipMemcpyDeviceToHost);
  
  wbLog(TRACE, "output is ");
  for (int i = 0; i < 20; i++)
  {
	   unsigned char temp = testingChar[i]
      wbLog(TRACE, hostInputImageData[i], " ", temp);
    
  }
  
 outputImage = wbImage_new(imageWidth, imageHeight, imageChannels, hostOutputImageData);
 wbSolution(args, outputImage);

  //@@ insert code here
  hipFree(cudaInputImageData);
  hipFree(cudaTemp2ImageData);
  free(hostInputImageData);
  free(testingChar);  
  
  return 0;

}

